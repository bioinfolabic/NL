#include "hip/hip_runtime.h"
extern "C" {
	#include "functions.h"
	#include "mt.h"
}

__constant__ int d_nmol;


/*=====================================================================================================*/
/***   Timer   ***/

/* Initialize Timer */
void initTimer() {
	t_i = time(NULL);
	gettimeofday(&last_t, NULL);
}

/* Finish Timer */
void finTimer() {
	delta_t = time(NULL) - t_i;
}

/* Subtract two values of time */
void timeval_subtract (struct timeval *result, struct timeval *x, struct timeval *y) {
	int nsec;

	/* Perform the carry for the later subtraction by updating y. */
	if (x->tv_usec < y->tv_usec) {
		nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
		y->tv_usec -= 1000000 * nsec;
		y->tv_sec += nsec;
	}
	if (x->tv_usec - y->tv_usec > 1000000) {
		nsec = (x->tv_usec - y->tv_usec) / 1000000;
		y->tv_usec += 1000000 * nsec;
		y->tv_sec -= nsec;
	}

	/* Compute the time remaining to wait. tv_usec is certainly positive. */
	result->tv_sec = x->tv_sec - y->tv_sec;
	result->tv_usec = x->tv_usec - y->tv_usec;
}



/*=====================================================================================================*/
/***   Files   ***/

/* Get Parameters */ 
/* Set function to get parameters */
int getParameter(const char *field_name, const char *token, const char *format, void *variable, FILE *fi) {
	char line[SEQ_MAX_LENGTH+12];
	char *p_token, *p_aux;
	int flag_done = 0;
	int len, i;

	do {
		if(fgets(line, SEQ_MAX_LENGTH+12, fi) == 0)
			return FAIL;

		len = strlen(line);
		if(line[len-1] == '\n')
			line[len-1] = '\0';

		switch(flag_done) {
			case 0:
				if(strstr(line, "/*") != 0) {
					flag_done = 1;
					if(strstr(line, "*/") != 0)
						flag_done = 2;
				}
				break;
			case 1:
				if(strstr(line, "*/") != 0)
					flag_done = 2;
				break;
			case 2:
				if(strstr(line, "/*") != 0) {
					flag_done = 1;
					if(strstr(line, "*/") != 0)
						flag_done = 2;
				} else {
					flag_done = 0;
				}
				break;
		}
	} while(flag_done != 0);

	if(strstr(line, field_name) != 0) {
		p_token = strtok(line, token);

		while(p_token != NULL) {
			p_aux = p_token;
			p_token = strtok(NULL, token);
		}

		if(p_aux[0] == ' ') {

			for(i = 0; i < len - 1; i++) {
				p_aux[i] = p_aux[i + 1];
			}
		}
		sscanf(p_aux, format, variable);
		return SUCCESS;

	} else {
		return FAIL;
	}
}

/* Load File */
/* Load input file and get parameters from there */
int loadFile(char **argv) {
	FILE *fi;
		fi = fopen(argv[1], "r"); // Need to indicate the file to execute

	if(fi != 0) {
		getParameter("sequence", "=", "%s", sequence, fi);
		getParameter("ProtLen", "=", "%d", &prot_len, fi);      
		getParameter("LV", "=", "%lf", &LV, fi);
		getParameter("stepLimit", "=", "%d", &step_limit, fi);
		getParameter("temperature", "=", "%lf", &temperature, fi);
		getParameter("savepathways", "=", "%c", &save_pathways, fi);
		getParameter("pathwaysstep", "=", "%d", &pathways_step, fi);

		fclose(fi);

		mass = 1.0; 
		n_mol = prot_len;
		bond_len = 1.0;
		n_c = (prot_len-1);
		dt = 0.0001;
		c_T = 0.01;
		display_interval = 100;
		step2resc_vels = 1;
		temperature_steps = 0.10;
		report_file = 'y';
		step2report = 'n';
		print_summary = 160;
		print_summary_interval = 'y';
		print_summary2file = 'y';
		shake_cons_prec = 1.0e-06;
		shake_max_cycle = 1; 
		shake_step2shake = 10;
		blockSize=BLOCKSIZE;
		blockSizeSum=BLOCKSIZESUM;
		n_blocks=(n_mol/blockSize);
		n_blocks_sum=(n_mol/blockSizeSum);

		if(n_mol%blockSize!=0)
			n_blocks++;
		if(n_mol%blockSizeSum!=0)
			n_blocks_sum++;



		if(print_summary != 'y' && print_summary != 'Y') print_summary_interval = step_limit - 1;
		if(report_file != 'y' && report_file != 'Y') step2report = step_limit - 1;
		if(save_pathways != 'y' && save_pathways != 'Y') pathways_step = step_limit - 1;
		en_update = min(print_summary_interval, step2report);
		en_update = min(en_update, pathways_step);

		return SUCCESS;
	} else {
		return FAIL;
	}
}

/* Put Parameters */
/* Print the parameters in the screen */
void putParameters() {
	printf("Sequence >>                         %s\n", sequence);
	printf("Mass >>                             %lf\n", mass);
	printf("Number of Particles >>              %d\n", n_mol);
	printf("Bond Length >>                      %d\n", bond_len);
	printf("Protein Length >>                   %d\n", prot_len);
	printf("Number of Constraints >>            %d\n", n_c);
	printf("Dimension of the Box >>             %lf\n", LV);
	printf("Time Step >>                        %lf\n", dt);
	printf("cT >>                               %lf\n", c_T);
	printf("Display Interval >>                 %d\n", display_interval);
	printf("Step Limit >>                       %d\n", step_limit);
	printf("Steps to Rescale Velocities >>      %d\n", step2resc_vels);
	printf("Temperature >>                      %lf\n", temperature);
	printf("Temperature Steps >>                %lf\n", temperature_steps);
	printf("Report File >>                      %c\n", report_file);
	printf("Steps to Report >>                  %d\n", step2report);
	printf("Print Summary >>                    %c\n", print_summary);
	printf("Print Summary Interval >>           %d\n", print_summary_interval);
	printf("Print Summary to File >>            %c\n", print_summary2file);
	printf("Shake-consPrec >>                   %lf\n", shake_cons_prec);
	printf("Shake-maxCycle >>                   %d\n", shake_max_cycle);
	printf("Steps to Shake >>                   %d\n", shake_step2shake);
	printf("Save Pathways >>                    %c\n", save_pathways);
	printf("Pathways Step >>                    %d\n", pathways_step);
}



/*=====================================================================================================*/
/***   Utilities (1/2)  ***/

/* Generate a random number double */
/* Mode: Mersenne Twister */
double randdouble(double max) {
	double ret;


	ret = fabs(((randomMT() + (RAND_MAX_MT/2)) / ((double)(RAND_MAX_MT)) * max));

	return ret;
}

/* Particle Position is Unique */
/* Verify if the particle position is unique */
int isUnique(Particle *p, int last) {
	int i;
	for(i = 0; i < last; i++) {
		if((p[i].v_r.x == p[last].v_r.x) && (p[i].v_r.y == p[last].v_r.y) && (p[i].v_r.z == p[last].v_r.z))
			return 0;
	}
	return 1;
}

/* Verify Boundary conditions*/
void verifyBoundary1(VectorR *v) {
	if(v->x < 0) {
		v->x += LV;
	} else {
		if(v->x >= LV)
			v->x -= LV;
	}

	if(v->y < 0) {
		v->y += LV;
	} else {
		if(v->y >= LV)
			v->y -= LV;
	}

	if(v->z < 0) {
		v->z += LV;
	} else {
		if(v->z >= LV)
			v->z -= LV;
	}
}

/* Verify Boundary conditions*/
void verifyBoundary2(VectorR *v) {
	if(v->x >= 0.5 * LV) {
		v->x -= LV;
	} else {
		if(v->x < -0.5 * LV)
			v->x += LV;
	}

	if(v->y >= 0.5 * LV) {
		v->y -= LV;
	} else {
		if(v->y < -0.5 * LV)
			v->y += LV;
	}

	if(v->z >= 0.5 * LV) {
		v->z -= LV;
	} else {
		if(v->z < -0.5 * LV)
			v->z += LV;
	}
}

/* Verify boundary conditions in GPU */
__device__ void cudaVerifyBoundary1(VectorR *v, double LV) {
	if(v->x < 0) {
		v->x += LV;
	} else {
		if(v->x >= LV)
			v->x -= LV;
	}

	if(v->y < 0) {
		v->y += LV;
	} else {
		if(v->y >= LV)
			v->y -= LV;
	}

	if(v->z < 0) {
		v->z += LV;
	} else {
		if(v->z >= LV)
			v->z -= LV;
	}
}

/* Verify boundary conditions in GPU */
__device__ void cudaVerifyBoundary2(VectorR *v, double LV) {
	if(v->x >= 0.5 * LV) {
		v->x -= LV;
	} else {
		if(v->x < -0.5 * LV)
			v->x += LV;
	}

	if(v->y >= 0.5 * LV) {
		v->y -= LV;
	} else {
		if(v->y < -0.5 * LV)
			v->y += LV;
	}

	if(v->z >= 0.5 * LV) {
		v->z -= LV;
	} else {
		if(v->z < -0.5 * LV)
			v->z += LV;
	}
}



/*=====================================================================================================*/
/***   Initialize   ***/

/* Alloc Arrays */
/* Alloc molecules and energies  */
void allocArrays() {
	particles = (Particle *) malloc(n_mol * sizeof(Particle));

	best_structure = (Particle *) malloc(n_mol * sizeof(Particle));

	constraint = (Constraint *) malloc(n_c * sizeof(Constraint)); 	

	auxSumLJ = (double *) malloc(n_mol * sizeof(double));

	auxSumB = (double *) malloc(n_mol * sizeof(double));

	auxSumT = (double *) malloc(n_mol * sizeof(double));

}

/* Alloc Device's variables */
/* Alloc GPU memory to molecules and energies */
void allocDevice() {
	hipMemcpyToSymbol(HIP_SYMBOL(d_nmol), &n_mol, sizeof(int));							
	hipMalloc((void **)&d_particles, n_blocks*blockSize * sizeof(Particle));
	hipMalloc((void **)&d_uB, sizeof(double));
	hipMalloc((void **)&d_uT, sizeof(double));
	hipMalloc((void **)&d_uLJ, sizeof(double));
	hipMalloc((void **)&d_uLJVector, n_blocks*blockSize * sizeof(double));	   
	hipMalloc((void **)&d_uBVector, n_blocks*blockSize * sizeof(double));	  	
	hipMalloc((void **)&d_uTVector, n_blocks*blockSize * sizeof(double));	  
	hipMalloc((void **)&d_auxSumLJ, n_blocks*blockSize * sizeof(double));
	hipMalloc((void **)&d_auxSumB, n_blocks*blockSize * sizeof(double));
	hipMalloc((void **)&d_auxSumT, n_blocks*blockSize * sizeof(double));
	hipMalloc((void **)&d_sequence, n_mol * sizeof(char));					  
	hipMalloc((void **)&d_constraint, n_c * sizeof(Constraint));
}

/* Setting velocity magnetude */
/* Set the velocity magnetude according to the temperature */
void setVelMag() {
	vel_mag = sqrt(N_DIM * (1. - 1./prot_len) * temperature);
}

/* Initialize Coordinates */
/* The molecular coordinates are initialized in a 3d lattice */
void initCoords() {
	int i, j, conf_OK;
	double theta;
	double phi;
	double dist;

	particles[0].v_r.x = LV/2; // First particle(Amino-acid)
	particles[0].v_r.y = LV/2;
	particles[0].v_r.z = LV/2;
	for(i = 1;i < n_mol; i++) {
		do {
			theta = randdouble(180.0) * M_PI / 180.0;
			phi = randdouble(360.0) * M_PI / 180.0;


			particles[i].v_r.x = particles[i-1].v_r.x + sin(theta) * cos(phi);
			particles[i].v_r.y = particles[i-1].v_r.y + sin(theta) * sin(phi);
			particles[i].v_r.z = particles[i-1].v_r.z + cos(theta);

			verifyBoundary1(&particles[i].v_r);

			conf_OK = 1; 
			for (j = 0; j < i; j++)
			{
				dist = sqrt(sqr(particles[i].v_r.x - particles[j].v_r.x) + sqr(particles[i].v_r.y - particles[j].v_r.y) + sqr(particles[i].v_r.z - particles[j].v_r.z));
				if (dist < 1)
					conf_OK = 0; 
			}
		} while((isUnique(particles, i) != 1) || (conf_OK != 1));
	}
}

/* Initialize Velocities */
/* The velocities are initialized with a magnetude dependent on the temperature (See function serVelMag()) */
/* The directions of the velocities are randomized */
void initVels() {
	int i;
	double theta;
	double phi;
	VectorR sum_v;

	sum_v.x = 0;
	sum_v.y = 0;
	sum_v.z = 0;

	/***   Generating Vectors   ***/
	setVelMag();
	for(i = 0; i < n_mol; i++) {
		theta = randdouble(180.0) * M_PI / 180.0;
		phi = randdouble(360.0) * M_PI / 360.0;

		particles[i].v_v.x = sin(theta) * cos(phi) * vel_mag;
		particles[i].v_v.y = sin(theta) * sin(phi) * vel_mag;
		particles[i].v_v.z = cos(theta) * vel_mag;

		sum_v.x += particles[i].v_v.x;
		sum_v.y += particles[i].v_v.y;
		sum_v.z += particles[i].v_v.z;

	}

	/***   Center of Mass at Rest   ***/
	for(i = 0; i < n_mol; i++) {
		particles[i].v_v.x = particles[i].v_v.x - (sum_v.x / prot_len);
		particles[i].v_v.y = particles[i].v_v.y - (sum_v.y / prot_len);
		particles[i].v_v.z = particles[i].v_v.z - (sum_v.z / prot_len);

	}
}

/* Initialize Accelerations */
/* The accelerations are initialized to zero */
void initAccs() {
	int i;

	for(i = 0; i < n_mol; i++) {
		particles[i].v_a.x = 0.;
		particles[i].v_a.y = 0.;
		particles[i].v_a.z = 0.;
	}
}

/* Build the constraint matrix */
void buildConstMatrix() {
	int i;

	for(i = 0; i < n_c; i++) {		// n_c= Number of constraints (Protein lenght - 1 )
		/***   Distance sqr taken as 1   ***/
		constraint[i].ik = i;
		constraint[i].jk = i + 1;

	}
}

/* Initialize MD simulation Parameters */
void initMD() {
	allocArrays();
	allocDevice();
	initCoords();
	initVels();
	initAccs();
	buildConstMatrix();
	hipMemcpy(d_sequence, sequence, n_mol * sizeof(char), hipMemcpyHostToDevice);

}



/*=====================================================================================================*/
/***   MD   ***/

/* Update the coordinates in GPU */
__global__ void cudaUpdatePos(Particle *particles, double dt ,double LV) {
	int b_Size=blockDim.x;
	int b_Id=blockIdx.x;

	int threadId=threadIdx.x+b_Id*b_Size;
	if(threadId<d_nmol){
		particles[threadId].v_r.x += particles[threadId].v_v.x * dt + 0.5 * particles[threadId].v_a.x * sqr(dt);
		particles[threadId].v_r.y += particles[threadId].v_v.y * dt + 0.5 * particles[threadId].v_a.y * sqr(dt);
		particles[threadId].v_r.z += particles[threadId].v_v.z * dt + 0.5 * particles[threadId].v_a.z * sqr(dt);

		/* Verify Boundary */
		if(particles[threadId].v_r.x < 0) {
			particles[threadId].v_r.x += LV;
		} else {
			if(particles[threadId].v_r.x >= LV)
				particles[threadId].v_r.x -= LV;
		}

		if(particles[threadId].v_r.y < 0) {
			particles[threadId].v_r.y += LV;
		} else {
			if(particles[threadId].v_r.y >= LV)
				particles[threadId].v_r.y -= LV;
		}

		if(particles[threadId].v_r.z < 0) {
			particles[threadId].v_r.z += LV;
		} else {
			if(particles[threadId].v_r.z >= LV)
				particles[threadId].v_r.z -= LV;
		}

		// First part of the velocity verlet
		particles[threadId].v_v.x += 0.5 * dt * particles[threadId].v_a.x;
		particles[threadId].v_v.y += 0.5 * dt * particles[threadId].v_a.y;
		particles[threadId].v_v.z += 0.5 * dt * particles[threadId].v_a.z;

		// Initialize the step acceleration
		particles[threadId].v_a.x = 0.;
		particles[threadId].v_a.y = 0.;
		particles[threadId].v_a.z = 0.;

	}
}

/* Compute the bond angle energy in GPU */
__global__ void cudaBondEnergy(Particle *particles, double *d_uBVector, double LV) {
	double d_uBond;
	int blockSize = blockDim.x;
	int threadId = threadIdx.x;
	int id= threadId+blockSize*blockIdx.x;
	VectorR dr1, dr2, a1, a2;
	double c11, c12, c22, cd, f;

		if(id < (d_nmol - 2)) {
			dr1.x = particles[id + 1].v_r.x - particles[id].v_r.x;
			dr1.y = particles[id + 1].v_r.y - particles[id].v_r.y;
			dr1.z = particles[id + 1].v_r.z - particles[id].v_r.z;
			dr2.x = particles[id + 2].v_r.x - particles[id + 1].v_r.x;
			dr2.y = particles[id + 2].v_r.y - particles[id + 1].v_r.y;
			dr2.z = particles[id + 2].v_r.z - particles[id + 1].v_r.z;

			cudaVerifyBoundary2(&dr1, LV);
			cudaVerifyBoundary2(&dr2, LV);
 
			c11 = dr1.x * dr1.x + dr1.y * dr1.y + dr1.z * dr1.z;
			c12 = dr1.x * dr2.x + dr1.y * dr2.y + dr1.z * dr2.z;
			c22 = dr2.x * dr2.x + dr2.y * dr2.y + dr2.z * dr2.z;

			cd = sqrt (c11 * c22);

			d_uBond = c12;

			f = -1.0;
			a1.x = (f / cd) * ((c12 / c11) * dr1.x  - dr2.x);
			a1.y = (f / cd) * ((c12 / c11) * dr1.y  - dr2.y);
			a1.z = (f / cd) * ((c12 / c11) * dr1.z  - dr2.z);
			a2.x = (f / cd) * (dr1.x - (c12 / c22) * dr2.x);
			a2.y = (f / cd) * (dr1.y - (c12 / c22) * dr2.y);
			a2.z = (f / cd) * (dr1.z - (c12 / c22) * dr2.z);



			atomicAdd(&particles[id].v_a.x, a1.x);
			atomicAdd(&particles[id].v_a.y, a1.y);
			atomicAdd(&particles[id].v_a.z, a1.z);
			atomicAdd(&particles[id + 1].v_a.x, -(a1.x + a2.x));
			atomicAdd(&particles[id + 1].v_a.y, -(a1.y + a2.y));
			atomicAdd(&particles[id + 1].v_a.z, -(a1.z + a2.z));
			atomicAdd(&particles[id + 2].v_a.x, a2.x);
			atomicAdd(&particles[id + 2].v_a.y, a2.y);
			atomicAdd(&particles[id + 2].v_a.z, a2.z);


		} else {
			d_uBond= 0;       
		}
		/* Bond angle potential */
		d_uBVector[id]=d_uBond;
}

/* Compute the torsion force energy in GPU */
__global__ void cudaTorsionEnergy(Particle *particles, double *d_uTVector, double LV) {
	double d_uTorsion;
	int blockSize = blockDim.x;
	int threadId = threadIdx.x;
	int id= threadId +blockSize*blockIdx.x;
	VectorR dr1, dr2, dr3, a1, a2;
	double c11, c12, c13, c22, c23, c33, pi, qia, qib, cr1, cr2, t1, t2, t3, t4, t5, t6, f;
	if (id<d_nmol){
			
		if(id < (d_nmol - 3)) {
			dr1.x = particles[id + 1].v_r.x - particles[id].v_r.x;
			dr1.y = particles[id + 1].v_r.y - particles[id].v_r.y;
			dr1.z = particles[id + 1].v_r.z - particles[id].v_r.z;
			dr2.x = particles[id + 2].v_r.x - particles[id + 1].v_r.x;
			dr2.y = particles[id + 2].v_r.y - particles[id + 1].v_r.y;
			dr2.z = particles[id + 2].v_r.z - particles[id + 1].v_r.z;
			dr3.x = particles[id + 3].v_r.x - particles[id + 2].v_r.x;
			dr3.y = particles[id + 3].v_r.y - particles[id + 2].v_r.y;
			dr3.z = particles[id + 3].v_r.z - particles[id + 2].v_r.z;

			cudaVerifyBoundary2(&dr1, LV);
			cudaVerifyBoundary2(&dr2, LV);
			cudaVerifyBoundary2(&dr3, LV);

			c11 = dr1.x * dr1.x + dr1.y * dr1.y + dr1.z * dr1.z;
			c12 = dr1.x * dr2.x + dr1.y * dr2.y + dr1.z * dr2.z;
			c13 = dr1.x * dr3.x + dr1.y * dr3.y + dr1.z * dr3.z;
			c22 = dr2.x * dr2.x + dr2.y * dr2.y + dr2.z * dr2.z;
			c23 = dr2.x * dr3.x + dr2.y * dr3.y + dr2.z * dr3.z;
			c33 = dr3.x * dr3.x + dr3.y * dr3.y + dr3.z * dr3.z;

			pi = c13 * c22 - c12 * c23;
			qia = c11 * c22 - c12 * c12;
			qib = c22 * c33 - c23 * c23;
			cr1 = c12 / c22;
			cr2 = c23 / c22;

			t1 = pi;
			t2 = c11 * c23 - c12 * c13;
			t3 = - qia;
			t4 = qib;
			t5 = c13 * c23 - c12 * c33;
			t6 = - t1;

			d_uTorsion = (- 0.5) * c13;

			f = 0.5;

			a1.x = f * c22 * (t1 * dr1.x + t2 * dr2.x + t3 * dr3.x) / (sqrt(qia * qib) * qia);
			a1.y = f * c22 * (t1 * dr1.y + t2 * dr2.y + t3 * dr3.y) / (sqrt(qia * qib) * qia);
			a1.z = f * c22 * (t1 * dr1.z + t2 * dr2.z + t3 * dr3.z) / (sqrt(qia * qib) * qia);
			a2.x = f * c22 * (t4 * dr1.x + t5 * dr2.x + t6 * dr3.x) / (sqrt(qia * qib) * qib);
			a2.y = f * c22 * (t4 * dr1.y + t5 * dr2.y + t6 * dr3.y) / (sqrt(qia * qib) * qib);
			a2.z = f * c22 * (t4 * dr1.z + t5 * dr2.z + t6 * dr3.z) / (sqrt(qia * qib) * qib);

			atomicAdd(&particles[id].v_a.x, a1.x);
			atomicAdd(&particles[id].v_a.y, a1.y);
			atomicAdd(&particles[id].v_a.z, a1.z);
			atomicAdd(&particles[id + 1].v_a.x, (- (1. + cr1) * a1.x) + cr2 * a2.x);
			atomicAdd(&particles[id + 1].v_a.y, (- (1. + cr1) * a1.y) + cr2 * a2.y);
			atomicAdd(&particles[id + 1].v_a.z, (- (1. + cr1) * a1.z) + cr2 * a2.z);
			atomicAdd(&particles[id + 2].v_a.x, cr1 * a1.x + (- (1. + cr2)) * a2.x);
			atomicAdd(&particles[id + 2].v_a.y, cr1 * a1.y + (- (1. + cr2)) * a2.y);
			atomicAdd(&particles[id + 2].v_a.z, cr1 * a1.z + (- (1. + cr2)) * a2.z);
			atomicAdd(&particles[id + 3].v_a.x, a2.x);
			atomicAdd(&particles[id + 3].v_a.y, a2.y);
			atomicAdd(&particles[id + 3].v_a.z, a2.z);
		} else {
			d_uTorsion = 0.;
		}
		
	}

	/* Torsion force potential */
	d_uTVector[id]=d_uTorsion;
}

/* Compute Lennard-Jones potential energy in GPU */
__global__ void cudaLJEnergy(Particle *particles, char *d_sequence, double LV, double *d_uLJVector, int n_mol) {

	double d_uLJComp=0.;
	int threadId = threadIdx.x;
	int blockSize = blockDim.x;
	int blockId= blockIdx.x;
	int i= threadId + blockId*blockSize;
	double r2, u_LJ, f_LJ;
	VectorR dr1;

	if (i<n_mol)
	{
		for(int j=(i+2); (j < n_mol) ;j++) {

				dr1.x = particles[i].v_r.x - particles[j].v_r.x;
				dr1.y = particles[i].v_r.y - particles[j].v_r.y;
				dr1.z = particles[i].v_r.z - particles[j].v_r.z;

				cudaVerifyBoundary2(&dr1, LV);

				r2 = (dr1.x * dr1.x) + (dr1.y * dr1.y) + (dr1.z * dr1.z);
				u_LJ = 4. * (pow(r2, -6) - pow(r2, -3));
				f_LJ = 24. * (2. * pow(r2, -7) - pow(r2, -4));


				//Iterations AB or BB		
				if( (d_sequence[i] != 'A') || (d_sequence[j] != 'A') ) {
					u_LJ = 0.5 * u_LJ;
					f_LJ = 0.5 * f_LJ;
				}

				atomicAdd(&particles[i].v_a.x, (dr1.x * f_LJ));
				atomicAdd(&particles[i].v_a.y, (dr1.y * f_LJ));
				atomicAdd(&particles[i].v_a.z, (dr1.z * f_LJ));
				atomicAdd(&particles[j].v_a.x, -(dr1.x * f_LJ));
				atomicAdd(&particles[j].v_a.y, -(dr1.y * f_LJ));
				atomicAdd(&particles[j].v_a.z, -(dr1.z * f_LJ));

				d_uLJComp += u_LJ;
				

			
		}  


	}
	/* Lennard-Jones potential energy */
	d_uLJVector[i] = d_uLJComp;

}

/* Update the velocities in GPU */
__global__ void cudaUpdateVelocities(Particle *particles, double dt) {
	int i=threadIdx.x + blockDim.x*blockIdx.x;

/* Second part of the velocity verlet */
	if(i<d_nmol){
		particles[i].v_v.x += 0.5 * dt * particles[i].v_a.x;
		particles[i].v_v.y += 0.5 * dt * particles[i].v_a.y;
		particles[i].v_v.z += 0.5 * dt * particles[i].v_a.z;
	}
}

/* Reduction of Sum */
/* Auxiliate the sum of vectors */
template <unsigned int blockSize>
__device__ void sumReduce(volatile double *sdata, unsigned int tid) {
	if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
	if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
	if (blockSize >=  16) sdata[tid] += sdata[tid +  8];
	if (blockSize >=   8) sdata[tid] += sdata[tid +  4];
	if (blockSize >=   4) sdata[tid] += sdata[tid +  2];
	if (blockSize >=   2) sdata[tid] += sdata[tid +  1];
}

/* Vector sum in GPU */
template <unsigned int blockSize>
__global__ void sumRise(double *g_idata, double *g_odata, unsigned int n) {
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;
	sdata[tid] = 0.;
	while (i < n)
		{
			if((i+blockSize)<n)
				sdata[tid] += g_idata[i] + g_idata[i+blockSize];
			else
				sdata[tid] += g_idata[i];
			i += gridSize;  
		}__syncthreads();
	
	if (blockSize >= 1024) 
		{ 
			if (tid < 512) 
				{ 
					sdata[tid] += sdata[tid + 512]; 
				} __syncthreads(); 
		}   
	if (blockSize >= 512) 
		{ 
			if (tid < 256) 
				{ 
					sdata[tid] += sdata[tid + 256]; 
				} __syncthreads(); 
		}
	
	if (blockSize >= 256) 
		{ 
			if (tid < 128) 
				{
					 sdata[tid] += sdata[tid + 128]; 
				} __syncthreads(); 
		}
	if (blockSize >= 128) 
		{ 
			if (tid <  64)
			 {
				 sdata[tid] += sdata[tid +  64]; 
			 } __syncthreads(); 
		}
	if (tid < 32)sumReduce<blockSize>(sdata, tid);
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

/* Step */
/* Function responsible for a single iteration */
/* Handle the calculation of moviments and energies */
void step() {

	hipMemcpy(d_particles, particles, n_mol * sizeof(Particle), hipMemcpyHostToDevice);

	/***   Update positions of all atoms   ***/
	cudaUpdatePos <<< n_blocks, blockSize >>> (d_particles, dt, LV);
	hipDeviceSynchronize();


	// Bond Energy
	cudaBondEnergy<<<n_blocks, blockSize>>>(d_particles,d_uBVector,LV);
	hipDeviceSynchronize();

	// Torsion Energy
	cudaTorsionEnergy<<<n_blocks, blockSize>>>(d_particles,d_uTVector,LV);
	hipDeviceSynchronize();

	//LJ Energy
	cudaLJEnergy <<<n_blocks, blockSize>>> (d_particles, d_sequence, LV,d_uLJVector, n_mol);
	hipDeviceSynchronize();

	cudaUpdateVelocities <<< n_blocks, blockSize >>> (d_particles, dt);
	hipDeviceSynchronize();

	// Sum of the Bond angle potential energy vector
	sumRise<BLOCKSIZESUM><<<n_blocks_sum,blockSizeSum,blockSizeSum*sizeof(double)>>>(d_uBVector,d_auxSumB,n_mol);
	hipDeviceSynchronize();
	sumRise<BLOCKSIZESUM><<<n_blocks_sum,blockSizeSum,blockSizeSum*sizeof(double)>>>(d_auxSumB,d_auxSumB,n_blocks_sum);
	hipDeviceSynchronize();

	// Sum of the Torsion forces potential energy vector
	sumRise<BLOCKSIZESUM><<<n_blocks_sum,blockSizeSum,blockSizeSum*sizeof(double)>>>(d_uTVector,d_auxSumT,n_mol);
	hipDeviceSynchronize();
	sumRise<BLOCKSIZESUM><<<n_blocks_sum,blockSizeSum,blockSizeSum*sizeof(double)>>>(d_auxSumT,d_auxSumT,n_blocks_sum);
	hipDeviceSynchronize();
	
	// Sum of the Lennard-Jones Potentia energy vector
	sumRise<BLOCKSIZESUM><<<n_blocks_sum,blockSizeSum,blockSizeSum*sizeof(double)>>>(d_uLJVector,d_auxSumLJ,n_mol);
	hipDeviceSynchronize();
	sumRise<BLOCKSIZESUM><<<n_blocks_sum,blockSizeSum,blockSizeSum*sizeof(double)>>>(d_auxSumLJ,d_auxSumLJ,n_blocks_sum);
	hipDeviceSynchronize();
	hipMemcpy(auxSumB, d_auxSumB, n_mol*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(auxSumT, d_auxSumT, n_mol*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(auxSumLJ, d_auxSumLJ, n_mol*sizeof(double), hipMemcpyDeviceToHost);


	uBond=auxSumB[0];
	uTorsion=auxSumT[0];
	uLJ=auxSumLJ[0];


	hipMemcpy(particles, d_particles, n_mol * sizeof(Particle), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
}



/*=====================================================================================================*/
/***   SHAKE   ***/
/* Compute shake relaxation */
/* The function handles the geometrics constraints computations */
void shakeRelaxation() {
	VectorR dr, dv;
	double cDev, cDevR, cDevV, g, ga;
	int changed, m, m1, m2, maxCycle, i;

	maxCycle = 200;
	cDevR = 0;
	cDevV = 0;


	for(i = 0; i < n_mol; i++) {
		nCycleR = 0;
		changed = 1;

		while(nCycleR < maxCycle && changed) {
			nCycleR++;
			changed = 0;
			cDev = 0;

			for(m = 0; m < n_c; m++) {
				m1 = constraint[m].ik;
				m2 = constraint[m].jk;

				dr.x = particles[m1].v_r.x - particles[m2].v_r.x;
				dr.y = particles[m1].v_r.y - particles[m2].v_r.y;
				dr.z = particles[m1].v_r.z - particles[m2].v_r.z;
				verifyBoundary2(&dr);

				g = (sqr(dr.x) + sqr(dr.y) + sqr(dr.z) - 1.) / 4.;
				ga = fabs(g);
				cDev = (cDev > ga) ? cDev: ga;

				if(ga > shake_cons_prec) {
					changed = 1;
					particles[m1].v_r.x -= (g * dr.x);
					particles[m1].v_r.y -= (g * dr.y);
					particles[m1].v_r.z -= (g * dr.z);

					particles[m2].v_r.x += (g * dr.x);
					particles[m2].v_r.y += (g * dr.y);
					particles[m2].v_r.z += (g * dr.z);

				}
			}
		}

		cDevR = (cDev > cDevR) ? cDev: cDevR;

		nCycleV = 0;
		changed = 1;

		while(nCycleV < maxCycle && changed) {
			nCycleV++;
			changed = 0;
			cDev = 0;
			for(m = 0; m < n_c; m++) {
				m1 = constraint[m].ik;
				m2 = constraint[m].jk;

				dr.x = particles[m1].v_r.x - particles[m2].v_r.x;
				dr.y = particles[m1].v_r.y - particles[m2].v_r.y;
				dr.z = particles[m1].v_r.z - particles[m2].v_r.z;

				verifyBoundary2(&dr);

				dv.x = particles[m1].v_v.x - particles[m2].v_v.x;
				dv.y = particles[m1].v_v.y - particles[m2].v_v.y;
				dv.z = particles[m1].v_v.z - particles[m2].v_v.z;

				g = ((dr.x * dv.x) + (dr.y * dv.y) + (dr.z * dv.z)) / 2.;
				ga = fabs(g);
				cDev = (cDev > ga) ? cDev: ga;

				if(ga > shake_cons_prec) {
			
					changed = 1;
					particles[m1].v_v.x -= (g * dr.x);
					particles[m1].v_v.y -= (g * dr.y);
					particles[m1].v_v.z -= (g * dr.z);
					particles[m2].v_v.x += (g * dr.x);
					particles[m2].v_v.y += (g * dr.y);
					particles[m2].v_v.z += (g * dr.z);
				}
			}
		}

		cDevV = (cDev > cDevV) ? cDev : cDevV;

	}
}



/*=====================================================================================================*/
/***   Thermostat   ***/

/* Compute berendsen thermostat */
/* The function handles the Temperature control */
void berendsenThermostat() {
	double sum, lambda, temp;
	int i;

	sum = 0;
	for(i = 0; i < n_mol; i++) {
		sum += sqr(particles[i].v_v.x) + sqr(particles[i].v_v.y) + sqr(particles[i].v_v.z);
	}

	temp = sum / (3. * (n_mol - 1));
	lambda = sqrt(1. + (dt / c_T) * (temperature / temp - 1.));

	for(i = 0; i < n_mol; i++) {
		particles[i].v_v.x *= lambda;
		particles[i].v_v.y *= lambda;
		particles[i].v_v.z *= lambda;
	}
}



/*=====================================================================================================*/
/***   Utilities (2/2)  ***/
/* Calculate status */
/* Calculate total potential energy, bond length average, Current temperature and density */
void calcStatus() {
	int i;
	double sum;

	sum = 0.;
	for(i = 0; i < n_mol; i++) {
		sum += sqr(particles[i].v_v.x) + sqr(particles[i].v_v.y) + sqr(particles[i].v_v.z);
	}

	kinetic_energy = 0.5 * sum;
	current_temperature = sum / (3. * (n_mol - 1));
	density = n_mol / (LV * sqr(LV));

	uSum = uTorsion + uBond + uLJ;
	total_energy = uSum + kinetic_energy;

	sum = 0.;
	for(i = 0; i < n_mol - 1; i++) {
		sum += sqrt(sqr(particles[i].v_r.x - particles[i + 1].v_r.x) + sqr(particles[i].v_r.y - particles[i + 1].v_r.y) + sqr(particles[i].v_r.z - particles[i + 1].v_r.z));
	}
	bond_avg = sum / (n_mol - 1);
}

/* Calculate RG */
/* Computate the Gyration Radius, Hydrophobic Gyration Radius and Hydrophilic Gyration Radius*/
void calcRG() {
	int i;
	int h = 0, p = 0;
	double x_avg = 0., y_avg = 0., z_avg = 0.;
	double x_avgH = 0., y_avgH = 0., z_avgH = 0.;
	double x_avgP = 0., y_avgP = 0., z_avgP = 0.;

	rGH = 0.;
	rGP = 0.;
	rG = 0.;
	for(i = 0; i < n_mol; i++) {
		x_avg += particles[i].v_r.x;
		y_avg += particles[i].v_r.y;
		z_avg += particles[i].v_r.z;

		if(sequence[i] == 'A') {
			x_avgH += particles[i].v_r.x;
			y_avgH += particles[i].v_r.y;
			z_avgH += particles[i].v_r.z;

			h++;
		} else {
			x_avgP += particles[i].v_r.x;
			y_avgP += particles[i].v_r.y;
			z_avgP += particles[i].v_r.z;

			p++;
		}
	}

	x_avg = x_avg / n_mol;
	y_avg = y_avg / n_mol;
	z_avg = z_avg / n_mol;

	x_avgH = x_avgH / h;
	y_avgH = y_avgH / h;
	z_avgH = z_avgH / h;

	x_avgP = x_avgP / p;
	y_avgP = y_avgP / p;
	z_avgP = z_avgP / p;

	for(i = 0; i < n_mol; i++) {
		rG += sqr(particles[i].v_r.x - x_avg) + sqr(particles[i].v_r.y - y_avg) + sqr(particles[i].v_r.z - z_avg);

		if(sequence[i] == 'A')
			rGH += sqr(particles[i].v_r.x - x_avgH) + sqr(particles[i].v_r.y - y_avgH) + sqr(particles[i].v_r.z - z_avgH);
		else
			rGP += sqr(particles[i].v_r.x - x_avgP) + sqr(particles[i].v_r.y - y_avgP) + sqr(particles[i].v_r.z - z_avgP);
	}

	rG = sqrt(rG / n_mol);
	rGH = sqrt(rGH / h);
	rGP = sqrt(rGP / p);
}

/* Evaluete the strucutre */
/* Save the characterist of the best_structre until then */
void evaluate() {
	int i;

	calcStatus();

	if(bond_avg <= 1.) {
		if(i_step == 0) {
			best_potencial_energy = uSum;
			best_step = i_step;
		} else {
			if(uSum < best_potencial_energy) {
				best_potencial_energy = uSum;
				best_step = i_step;

				for(i = 0; i < n_mol; i++) {
					best_structure[i].v_r.x = particles[i].v_r.x;
					best_structure[i].v_r.y = particles[i].v_r.y;
					best_structure[i].v_r.z = particles[i].v_r.z;
				}
				calcRG();

			}
		}
	}

}

/* Calculate center of mass of the structure */
void calcCenterMass() {
	int i;
	VectorR r;

	r.x = 0.;
	r.y = 0.;
	r.z = 0.;
	for(i = 0; i < n_mol; i++) {
		r.x += particles[i].v_r.x;
		r.y += particles[i].v_r.y;
		r.z += particles[i].v_r.z;
	}

	center_mass.x = r.x / n_mol;
	center_mass.y = r.y / n_mol;
	center_mass.z = r.z / n_mol;
}

/* Print Summary */
void printSummary(char **argv) {
	FILE *fo;
	char file_name[200];


	calcCenterMass();
	system("clear");
	printf("Step: %d\n\nTemp = %lf\nTotal Lennard-Jones Potential = %lf\nTotal Torsion Potential = %lf\nTotal Chain Angle Potential = %lf\nTotal Potential Energy = %lf\n", i_step, current_temperature, uLJ, uTorsion, uBond, uSum);
	printf("Center of Mass = (%.2lf, %.2lf, %.2lf)\n", center_mass.x, center_mass.y, center_mass.z);
	printf("Radius of Gyration - Hydrophobic = %lf\nRadius of Gyration - Polar = %lf\nRadius of Gyration - All = %lf\n", rGH, rGP, rG);
	printf("Bond Length Average = %lf\n", bond_avg);

	if(print_summary2file == 'y' || print_summary2file == 'Y') {
		sprintf(file_name, "%s_summary_%s.txt", argv[2], argv[3]);
		fo = fopen(file_name, "a+");
		if(i_step == 0)
			fprintf(fo, "Step\tTemperature\tU_LJ\tU_Torsion\tU_ChainAngles\tU_Total\trGH\trGP\trG\n");

		fprintf(fo, "%d\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\n", i_step, temperature, uLJ, uTorsion, uBond, uSum, rGH, rGP, rG);
		fclose(fo);
	}
}

/* Save Pathways */
/* save the actual structure in a file */
void savePathways(char **argv) {
	FILE *fo;
	int i;
	char file_name[200];


	sprintf(file_name, "%s_%s.txt", argv[2], argv[3]);
	fo = fopen(file_name, "a+");

	fprintf(fo, "N\tx\ty\tz\n");
	for (i = 0; i < n_mol; i++) {
		fprintf(fo, "%d\t%lf\t%lf\t%lf\n", i, particles[i].v_r.x, particles[i].v_r.y, particles[i].v_r.z);
	}

	fprintf(fo, "\n\nPotential Energy = %lf\nStep = %d\n", uSum, i_step);
	fprintf(fo, "uLJ = %lf\nTorsion = %lf\nBond = %lf\n",  uLJ, uTorsion, uBond); 
	fprintf(fo, "rGAll = %lf\nrGH = %lf\nrGP = %lf\n\n\n", rG, rGH, rGP);

	fclose(fo);

}



/*=====================================================================================================*/
/***   End   ***/

/* Free arrays */
void freeArrays() {
	free(particles);
	free(best_structure);
	free(constraint);

}

/* Free device arrays and variables*/
void freeDevice() {
	hipFree(d_particles);
	hipFree(d_sequence);
	hipFree(d_uB);
	hipFree(d_uT);
	hipFree(d_uLJ);
	hipFree(d_uLJVector);
}

/* Finish */
/* Call the functions to end the program */
void finishSim(char **argv) {
	finTimer();
	savePathways(argv);
	freeArrays();
	freeDevice();
}
/*=====================================================================================================*/
